
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

int main (int argc, char* argv[]) {
  //TODO: add usage
  
  int n = atoi(argv[1]); //TODO: atoi is an unsafe function

  float* in1 = new float[n];
  float* in2 = new float[n];
  float* out = new float[n];
  
  
  for (int i=0; i<n; ++i) {
    in1[i] = i;
    in2[i] = 100+i;
  }


  
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();

  //gpu stuff goes here.

  
  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin);

  std::cout<<n<<" "<<totaltime.count()<<std::endl;

  delete[] in1;
  delete[] in2;
  delete[] out;

  return 0;
}
